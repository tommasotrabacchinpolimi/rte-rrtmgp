#include "hip/hip_runtime.h"
#include <chrono>
#include <functional>
#include <iostream>
#include <iomanip>

#include "rrtmgp_kernel_launcher_cuda.h"
#include "tools_gpu.h"
#include "Array.h"
#include "tuner.h"

#include <chrono>



namespace
{
    #include "gas_optics_kernels.cu"
}


namespace rrtmgp_kernel_launcher_cuda
{
    void reorder123x321(
            const int ni, const int nj, const int nk,
            const Float* arr_in, Float* arr_out)
    {
        Tuner_map& tunings = Tuner::get_map();

        dim3 grid(ni, nj, nk);
        dim3 block;

        if (tunings.count("reorder123x321_kernel") == 0)
        {
            std::tie(grid, block) = tune_kernel(
                "reorder123x321_kernel",
                dim3(ni, nj, nk),
                {1, 2, 4, 8, 16, 24, 32, 48, 64, 96},
                {1, 2, 4, 8, 16, 24, 32, 48, 64, 96},
                {1, 2, 4, 8, 16, 24, 32, 48, 64, 96},
                reorder123x321_kernel,
                ni, nj, nk, arr_in, arr_out);

            tunings["reorder123x321_kernel"].first = grid;
            tunings["reorder123x321_kernel"].second = block;
        }
        else
        {
            grid = tunings["reorder123x321_kernel"].first;
            block = tunings["reorder123x321_kernel"].second;
        }

        reorder123x321_kernel<<<grid, block>>>(
                ni, nj, nk, arr_in, arr_out);
    }


    void reorder12x21(
            const int ni, const int nj,
            const Float* arr_in, Float* arr_out)
    {
        const int block_i = 32;
        const int block_j = 16;

        const int grid_i = ni/block_i + (ni%block_i > 0);
        const int grid_j = nj/block_j + (nj%block_j > 0);

        dim3 grid_gpu(grid_i, grid_j);
        dim3 block_gpu(block_i, block_j);

        reorder12x21_kernel<<<grid_gpu, block_gpu>>>(
                ni, nj, arr_in, arr_out);
    }


    void zero_array(const int ni, const int nj, const int nk, Float* arr)
    {
        const int block_i = 32;
        const int block_j = 16;
        const int block_k = 1;

        const int grid_i = ni/block_i + (ni%block_i > 0);
        const int grid_j = nj/block_j + (nj%block_j > 0);
        const int grid_k = nk/block_k + (nk%block_k > 0);

        dim3 grid_gpu(grid_i, grid_j, grid_k);
        dim3 block_gpu(block_i, block_j, block_k);

        zero_array_kernel<<<grid_gpu, block_gpu>>>(
                ni, nj, nk, arr);

    }


    void interpolation(
            const int ncol, const int nlay,
            const int ngas, const int nflav, const int neta, const int npres, const int ntemp,
            const int* flavor,
            const Float* press_ref_log,
            const Float* temp_ref,
            Float press_ref_log_delta,
            Float temp_ref_min,
            Float temp_ref_delta,
            Float press_ref_trop_log,
            const Float* vmr_ref,
            const Float* play,
            const Float* tlay,
            Float* col_gas,
            int* jtemp,
            Float* fmajor, Float* fminor,
            Float* col_mix,
            Bool* tropo,
            int* jeta,
            int* jpress)
    {
        const int block_col  = 4;
        const int block_lay  = 2;
        const int block_flav = 16;

        const int grid_col  = ncol /block_col  + (ncol%block_col   > 0);
        const int grid_lay  = nlay /block_lay  + (nlay%block_lay   > 0);
        const int grid_flav = nflav/block_flav + (nflav%block_flav > 0);

        dim3 grid_gpu(grid_col, grid_lay, grid_flav);
        dim3 block_gpu(block_col, block_lay, block_flav);

        Float tmin = std::numeric_limits<Float>::min();
        interpolation_kernel<<<grid_gpu, block_gpu>>>(
                ncol, nlay, ngas, nflav, neta, npres, ntemp, tmin,
                flavor, press_ref_log, temp_ref,
                press_ref_log_delta, temp_ref_min,
                temp_ref_delta, press_ref_trop_log,
                vmr_ref, play, tlay,
                col_gas, jtemp, fmajor,
                fminor, col_mix, tropo,
                jeta, jpress);
    }


    void combine_abs_and_rayleigh(
            const int ncol, const int nlay, const int ngpt,
            const Float* tau_abs, const Float* tau_rayleigh,
            Float* tau, Float* ssa, Float* g)
    {
        Tuner_map& tunings = Tuner::get_map();

        Float tmin = std::numeric_limits<Float>::min();

        dim3 grid(ncol, nlay, ngpt);
        dim3 block;

        if (tunings.count("combine_abs_and_rayleigh_kernel") == 0)
        {
            std::tie(grid, block) = tune_kernel(
                "combine_abs_and_rayleigh_kernel",
                dim3(ncol, nlay, ngpt),
                {1, 2, 4, 8, 16, 24, 32, 48, 64, 96}, {1, 2, 4}, {1, 2, 4, 8, 16, 24, 32, 48, 64, 96},
                combine_abs_and_rayleigh_kernel,
                ncol, nlay, ngpt, tmin,
                tau_abs, tau_rayleigh,
                tau, ssa, g);

            tunings["combine_abs_and_rayleigh_kernel"].first = grid;
            tunings["combine_abs_and_rayleigh_kernel"].second = block;
        }
        else
        {
            grid = tunings["combine_abs_and_rayleigh_kernel"].first;
            block = tunings["combine_abs_and_rayleigh_kernel"].second;
        }

        combine_abs_and_rayleigh_kernel<<<grid, block>>>(
                ncol, nlay, ngpt, tmin,
                tau_abs, tau_rayleigh,
                tau, ssa, g);
    }


    void compute_tau_rayleigh(
            const int ncol, const int nlay, const int nbnd, const int ngpt,
            const int ngas, const int nflav, const int neta, const int npres, const int ntemp,
            const int* gpoint_flavor,
            const int* gpoint_bands,
            const int* band_lims_gpt,
            const Float* krayl,
            int idx_h2o, const Float* col_dry, const Float* col_gas,
            const Float* fminor, const int* jeta,
            const Bool* tropo, const int* jtemp,
            Float* tau_rayleigh)
    {
        Tuner_map& tunings = Tuner::get_map();

        dim3 grid(ncol, nlay, ngpt);
        dim3 block;

        if (tunings.count("compute_tau_rayleigh_kernel") == 0)
        {
            std::tie(grid, block) = tune_kernel(
                "compute_tau_rayleigh_kernel",
                dim3(ncol, nlay, ngpt),
                {1, 2, 4, 16, 24, 32}, {1, 2, 4}, {1, 2, 4, 8, 16},
                compute_tau_rayleigh_kernel,
                ncol, nlay, nbnd, ngpt,
                ngas, nflav, neta, npres, ntemp,
                gpoint_flavor,
                gpoint_bands,
                band_lims_gpt,
                krayl,
                idx_h2o, col_dry, col_gas,
                fminor, jeta,
                tropo, jtemp,
                tau_rayleigh);

            tunings["compute_tau_rayleigh_kernel"].first = grid;
            tunings["compute_tau_rayleigh_kernel"].second = block;
        }
        else
        {
            grid = tunings["compute_tau_rayleigh_kernel"].first;
            block = tunings["compute_tau_rayleigh_kernel"].second;
        }

        compute_tau_rayleigh_kernel<<<grid, block>>>(
                ncol, nlay, nbnd, ngpt,
                ngas, nflav, neta, npres, ntemp,
                gpoint_flavor,
                gpoint_bands,
                band_lims_gpt,
                krayl,
                idx_h2o, col_dry, col_gas,
                fminor, jeta,
                tropo, jtemp,
                tau_rayleigh);
    }


    struct Gas_optical_depths_major_kernel
    {
        template<unsigned int I, unsigned int J, unsigned int K, class... Args>
        static void launch(dim3 grid, dim3 block, Args... args)
        {
            gas_optical_depths_major_kernel<I, J, K><<<grid, block>>>(args...);
        }
    };


    struct Gas_optical_depths_minor_kernel
    {
        template<unsigned int I, unsigned int J, unsigned int K, class... Args>
        static void launch(dim3 grid, dim3 block, Args... args)
        {
            gas_optical_depths_minor_kernel<I, J, K><<<grid, block>>>(args...);
        }
    };



#ifndef SERIAL
    void compute_tau_absorption(
            const int ncol, const int nlay, const int nband, const int ngpt,
            const int ngas, const int nflav, const int neta, const int npres, const int ntemp,
            const int nminorlower, const int nminorklower,
            const int nminorupper, const int nminorkupper,
            const int idx_h2o,
            const int* gpoint_flavor,
            const int* band_lims_gpt,
            const Float* kmajor,
            const Float* kminor_lower,
            const Float* kminor_upper,
            const int* minor_limits_gpt_lower,
            const int* minor_limits_gpt_upper,
            const Bool* minor_scales_with_density_lower,
            const Bool* minor_scales_with_density_upper,
            const Bool* scale_by_complement_lower,
            const Bool* scale_by_complement_upper,
            const int* idx_minor_lower,
            const int* idx_minor_upper,
            const int* idx_minor_scaling_lower,
            const int* idx_minor_scaling_upper,
            const int* kminor_start_lower,
            const int* kminor_start_upper,
            const Bool* tropo,
            const Float* col_mix, const Float* fmajor,
            const Float* fminor, const Float* play,
            const Float* tlay, const Float* col_gas,
            const int* jeta, const int* jtemp,
            const int* jpress,
            Float* tau)
    {

        int* prova = (int*) malloc(sizeof(int) * 1);
        hipError_t err = hipMemcpy(prova, gpoint_flavor, sizeof(int) * 1, hipMemcpyDeviceToHost);
        if (err != hipSuccess) {
            std::cout << err << std::endl;
            std::cout << "execution failed" << std::endl;
            exit(1);
        }
        Tuner_map& tunings = Tuner::get_map();

        dim3 grid_gpu_maj(ngpt, nlay, ncol);
        dim3 block_gpu_maj;

        if (tunings.count("gas_optical_depths_major_kernel") == 0)
        {
            Float* tau_tmp = Tools_gpu::allocate_gpu<Float>(ngpt*nlay*ncol);
            std::tie(grid_gpu_maj, block_gpu_maj) =
                tune_kernel_compile_time<Gas_optical_depths_major_kernel>(
                    "gas_optical_depths_major_kernel",
                    dim3(ngpt, nlay, ncol),
                    std::integer_sequence<unsigned int, 1, 2, 4, 8, 16, 24, 32, 48, 64>{},
                    std::integer_sequence<unsigned int, 1, 2, 4>{},
                    std::integer_sequence<unsigned int, 8, 16, 24, 32, 48, 64, 96, 128, 256>{},
                    ncol, nlay, nband, ngpt,
                    nflav, neta, npres, ntemp,
                    gpoint_flavor, band_lims_gpt,
                    kmajor, col_mix, fmajor, jeta,
                    tropo, jtemp, jpress,
                    tau_tmp);

            Tools_gpu::free_gpu<Float>(tau_tmp);

            tunings["gas_optical_depths_major_kernel"].first = grid_gpu_maj;
            tunings["gas_optical_depths_major_kernel"].second = block_gpu_maj;
        }
        else
        {
            grid_gpu_maj = tunings["gas_optical_depths_major_kernel"].first;
            block_gpu_maj = tunings["gas_optical_depths_major_kernel"].second;
        }

        run_kernel_compile_time<Gas_optical_depths_major_kernel>(
                std::integer_sequence<unsigned int, 1, 2, 4, 8, 16, 24, 32, 48, 64>{},
                std::integer_sequence<unsigned int, 1, 2, 4>{},
                std::integer_sequence<unsigned int, 8, 16, 24, 32, 48, 64, 96, 128, 256>{},
                grid_gpu_maj, block_gpu_maj,
                ncol, nlay, nband, ngpt,
                nflav, neta, npres, ntemp,
                gpoint_flavor, band_lims_gpt,
                kmajor, col_mix, fmajor, jeta,
                tropo, jtemp, jpress,
                tau);

        // Lower
        int idx_tropo = 1;

        dim3 grid_gpu_min_1(1, 42, 8);
        dim3 block_gpu_min_1(8,1,16);
        #ifdef PROFILE
            float delta;
            hipEvent_t start, stop;
            hipEventCreate(&start);
            hipEventCreate(&stop);
            hipEventRecord(start, 0);
        #endif
        gas_optical_depths_minor_kernel<8,1,16><<<grid_gpu_min_1, block_gpu_min_1>>>(
                                        ncol, nlay, ngpt,
                                        ngas, nflav, ntemp, neta,
                                        nminorlower,
                                        nminorklower,
                                        idx_h2o, idx_tropo,
                                        gpoint_flavor,
                                        kminor_lower,
                                        minor_limits_gpt_lower,
                                        minor_scales_with_density_lower,
                                        scale_by_complement_lower,
                                        idx_minor_lower,
                                        idx_minor_scaling_lower,
                                        kminor_start_lower,
                                        play, tlay, col_gas,
                                        fminor, jeta, jtemp,
                                        tropo, tau, nullptr);

        #ifdef PROFILE
            hipEventRecord(stop, 0);
            hipEventSynchronize(stop);
            hipEventElapsedTime(&delta, start, stop);
            std::cout << "elapsed time for cuda kernel (lower): " << delta << std::endl;
        #endif

        // Upper
        idx_tropo = 0;

        dim3 grid_gpu_min_2(1, 42, 4);
        dim3 block_gpu_min_2(8,1,32);
        #ifdef PROFILE
            hipEventRecord(start, 0);
        #endif
        gas_optical_depths_minor_kernel<8,1,32><<<grid_gpu_min_2, block_gpu_min_2>>>(
                                    ncol, nlay, ngpt,
                                    ngas, nflav, ntemp, neta,
                                    nminorupper,
                                    nminorkupper,
                                    idx_h2o, idx_tropo,
                                    gpoint_flavor,
                                    kminor_upper,
                                    minor_limits_gpt_upper,
                                    minor_scales_with_density_upper,
                                    scale_by_complement_upper,
                                    idx_minor_upper,
                                    idx_minor_scaling_upper,
                                    kminor_start_upper,
                                    play, tlay, col_gas,
                                    fminor, jeta, jtemp,
                                    tropo, tau, nullptr);

        #ifdef PROFILE
            hipEventRecord(stop, 0);
            hipEventSynchronize(stop);
            hipEventElapsedTime(&delta, start, stop);
            std::cout << "elapsed time for cuda kernel (upper): " << delta << std::endl;
        #endif

    }

#else

    void* copy_from_gpu(void* source, int size) {
        void* ret = (void*) malloc(size);
        hipError_t err = hipMemcpy(ret, source, size, hipMemcpyDeviceToHost);
        if (err != hipSuccess) {
            std::cout << err << std::endl;
            std::cout << "execution failed" << std::endl;
            exit(1);
        }
        return ret;
    }

    void copy_to_gpu(void* source, void* dest, int size) {
        hipError_t err = hipMemcpy(dest, source, size, hipMemcpyHostToDevice);
        if (err != hipSuccess) {
            std::cout << err << std::endl;
            std::cout << "execution failed" << std::endl;
            exit(1);
        }
    }

    void compute_tau_absorption(
            const int ncol, const int nlay, const int nband, const int ngpt,
            const int ngas, const int nflav, const int neta, const int npres, const int ntemp,
            const int nminorlower, const int nminorklower,
            const int nminorupper, const int nminorkupper,
            const int idx_h2o,
            const int* gpoint_flavor,
            const int* band_lims_gpt,
            const Float* kmajor,
            const Float* kminor_lower,
            const Float* kminor_upper,
            const int* minor_limits_gpt_lower,
            const int* minor_limits_gpt_upper,
            const Bool* minor_scales_with_density_lower,
            const Bool* minor_scales_with_density_upper,
            const Bool* scale_by_complement_lower,
            const Bool* scale_by_complement_upper,
            const int* idx_minor_lower,
            const int* idx_minor_upper,
            const int* idx_minor_scaling_lower,
            const int* idx_minor_scaling_upper,
            const int* kminor_start_lower,
            const int* kminor_start_upper,
            const Bool* tropo,
            const Float* col_mix, const Float* fmajor,
            const Float* fminor, const Float* play,
            const Float* tlay, const Float* col_gas,
            const int* jeta, const int* jtemp,
            const int* jpress,
            Float* tau)
    {



        Tuner_map& tunings = Tuner::get_map();

        dim3 grid_gpu_maj(ngpt, nlay, ncol);
        dim3 block_gpu_maj;

        if (tunings.count("gas_optical_depths_major_kernel") == 0)
        {
            Float* tau_tmp = Tools_gpu::allocate_gpu<Float>(ngpt*nlay*ncol);
            std::tie(grid_gpu_maj, block_gpu_maj) =
               tune_kernel_compile_time<Gas_optical_depths_major_kernel>(
                    "gas_optical_depths_major_kernel",
                    dim3(ngpt, nlay, ncol),
                    std::integer_sequence<unsigned int, 1, 2, 4, 8, 16, 24, 32, 48, 64>{},
                    std::integer_sequence<unsigned int, 1, 2, 4>{},
                    std::integer_sequence<unsigned int, 8, 16, 24, 32, 48, 64, 96, 128, 256>{},
                    ncol, nlay, nband, ngpt,
                    nflav, neta, npres, ntemp,
                    gpoint_flavor, band_lims_gpt,
                    kmajor, col_mix, fmajor, jeta,
                    tropo, jtemp, jpress,
                    tau_tmp);

            Tools_gpu::free_gpu<Float>(tau_tmp);

            tunings["gas_optical_depths_major_kernel"].first = grid_gpu_maj;
            tunings["gas_optical_depths_major_kernel"].second = block_gpu_maj;
        }
        else
        {
            grid_gpu_maj = tunings["gas_optical_depths_major_kernel"].first;
            block_gpu_maj = tunings["gas_optical_depths_major_kernel"].second;
        }

        run_kernel_compile_time<Gas_optical_depths_major_kernel>(
                std::integer_sequence<unsigned int, 1, 2, 4, 8, 16, 24, 32, 48, 64>{},
                std::integer_sequence<unsigned int, 1, 2, 4>{},
                std::integer_sequence<unsigned int, 8, 16, 24, 32, 48, 64, 96, 128, 256>{},
                grid_gpu_maj, block_gpu_maj,
                ncol, nlay, nband, ngpt,
                nflav, neta, npres, ntemp,
                gpoint_flavor, band_lims_gpt,
                kmajor, col_mix, fmajor, jeta,
                tropo, jtemp, jpress,
                tau);

        // Lower
        int idx_tropo = 1;




        const int* gpoint_flavor_cpu = (const int*)copy_from_gpu((int*)gpoint_flavor, sizeof(int) * 2 * ngpt);
        const Float* kminor_lower_cpu = (const Float*)copy_from_gpu((Float*)kminor_lower, sizeof(Float) * ntemp * neta * nminorklower);
        const Float* kminor_upper_cpu = (const Float*)copy_from_gpu((Float*)kminor_upper, sizeof(Float) * ntemp * neta * nminorkupper);
        const int* minor_limits_gpt_lower_cpu = (const int*)copy_from_gpu((int*)minor_limits_gpt_lower, sizeof(int) * 2 * nminorlower);
        const int* minor_limits_gpt_upper_cpu = (const int*)copy_from_gpu((int*)minor_limits_gpt_upper, sizeof(int) * 2 * nminorupper);
        const Bool* minor_scales_with_density_lower_cpu = (const Bool*)copy_from_gpu((Bool*)minor_scales_with_density_lower, sizeof(Bool) * nminorlower);
        const Bool* minor_scales_with_density_upper_cpu = (const Bool*)copy_from_gpu((Bool*)minor_scales_with_density_upper, sizeof(Bool) * nminorupper);
        const Bool* scale_by_complement_lower_cpu = (const Bool*)copy_from_gpu((Bool*)scale_by_complement_lower, sizeof(Bool) * nminorlower);
        const Bool* scale_by_complement_upper_cpu = (const Bool*)copy_from_gpu((Bool*)scale_by_complement_upper, sizeof(Bool) * nminorupper);
        const int* idx_minor_lower_cpu = (const int*)copy_from_gpu((int*)idx_minor_lower, sizeof(int) * nminorlower);
        const int* idx_minor_upper_cpu = (const int*)copy_from_gpu((int*)idx_minor_upper, sizeof(int) * nminorupper);
        const int* idx_minor_scaling_lower_cpu = (const int*)copy_from_gpu((int*)idx_minor_scaling_lower, sizeof(int) * nminorlower);
        const int* idx_minor_scaling_upper_cpu = (const int*)copy_from_gpu((int*)idx_minor_scaling_upper, sizeof(int) * nminorupper);
        const int* kminor_start_lower_cpu = (const int*)copy_from_gpu((int*)kminor_start_lower, sizeof(int) * nminorlower);
        const int* kminor_start_upper_cpu = (const int*)copy_from_gpu((int*)kminor_start_upper, sizeof(int) * nminorupper);
        const Float* play_cpu = (const Float*)copy_from_gpu((Float*)play, sizeof(Float) * ncol * nlay);
        const Float* tlay_cpu = (const Float*)copy_from_gpu((Float*)tlay, sizeof(Float) * ncol * nlay);
        const Float* col_gas_cpu = (const Float*)copy_from_gpu((Float*)col_gas, sizeof(Float) * ncol * nlay * (ngas + 1));
        const Float* fminor_cpu = (const Float*)copy_from_gpu((Float*)fminor, sizeof(Float) * 2 * 2 * ncol * nlay * nflav);
        const int* jeta_cpu = (const int*)copy_from_gpu((int*)jeta, sizeof(int) * 2 * ncol * nlay * nflav);
        const int* jtemp_cpu = (const int*)copy_from_gpu((int*)jtemp, sizeof(int) * ncol * nlay);
        const Bool* tropo_cpu = (const Bool*)copy_from_gpu((Bool*)tropo, sizeof(Bool) * ncol * nlay);
        Float* tau_cpu = (Float*)copy_from_gpu((Float*)tau, sizeof(Float) * ncol * nlay * ngpt);

#ifdef PROFILE
        auto start = std::chrono::high_resolution_clock::now();
#endif
        gas_optical_depths_minor_serial<1,42,8,8,1,16>(ncol, nlay, ngpt,
                                        ngas, nflav, ntemp, neta,
                                        nminorlower,
                                        nminorklower,
                                        idx_h2o, idx_tropo,
                                        gpoint_flavor_cpu,
                                        kminor_lower_cpu,
                                        minor_limits_gpt_lower_cpu,
                                        minor_scales_with_density_lower_cpu,
                                        scale_by_complement_lower_cpu,
                                        idx_minor_lower_cpu,
                                        idx_minor_scaling_lower_cpu,
                                        kminor_start_lower_cpu,
                                        play_cpu, tlay_cpu, col_gas_cpu,
                                        fminor_cpu, jeta_cpu, jtemp_cpu,
                                        tropo_cpu, tau_cpu, nullptr);
#ifdef PROFILE
        auto stop = std::chrono::high_resolution_clock::now();
        std::cout << "elapsed timer for seriel kernel (lower): " << std::chrono::duration_cast<std::chrono::microseconds>(stop - start).count() / 1000.0 << std::endl;
#endif
        // Upper
        idx_tropo = 0;


#ifdef PROFILE
    start = std::chrono::high_resolution_clock::now();
#endif
        gas_optical_depths_minor_serial<1,42,4,8,1,32>(ncol, nlay, ngpt,
                                    ngas, nflav, ntemp, neta,
                                    nminorupper,
                                    nminorkupper,
                                    idx_h2o, idx_tropo,
                                    gpoint_flavor_cpu,
                                    kminor_upper_cpu,
                                    minor_limits_gpt_upper_cpu,
                                    minor_scales_with_density_upper_cpu,
                                    scale_by_complement_upper_cpu,
                                    idx_minor_upper_cpu,
                                    idx_minor_scaling_upper_cpu,
                                    kminor_start_upper_cpu,
                                    play_cpu, tlay_cpu, col_gas_cpu,
                                    fminor_cpu, jeta_cpu, jtemp_cpu,
                                    tropo_cpu, tau_cpu, nullptr);
#ifdef PROFILE
    stop = std::chrono::high_resolution_clock::now();
    std::cout << "elapsed timer for serial kernel (lower): " << std::chrono::duration_cast<std::chrono::microseconds>(stop - start).count()  / 1000.0 << std::endl;
#endif

        copy_to_gpu(tau_cpu, tau, sizeof(Float) * ncol * nlay * ngpt);

        free((void*)gpoint_flavor_cpu);
        free((void*)kminor_lower_cpu);
        free((void*)kminor_upper_cpu);
        free((void*)minor_limits_gpt_lower_cpu);
        free((void*)minor_limits_gpt_upper_cpu);
        free((void*)minor_scales_with_density_lower_cpu);
        free((void*)minor_scales_with_density_upper_cpu);
        free((void*)scale_by_complement_lower_cpu);
        free((void*)scale_by_complement_upper_cpu);
        free((void*)idx_minor_lower_cpu);
        free((void*)idx_minor_upper_cpu);
        free((void*)idx_minor_scaling_lower_cpu);
        free((void*)idx_minor_scaling_upper_cpu);
        free((void*)kminor_start_lower_cpu);
        free((void*)kminor_start_upper_cpu);
        free((void*)play_cpu);
        free((void*)tlay_cpu);
        free((void*)col_gas_cpu);
        free((void*)fminor_cpu);
        free((void*)jeta_cpu);
        free((void*)jtemp_cpu);
        free((void*)tropo_cpu);
        free((void*)tau_cpu);
    }
#endif

    void Planck_source(
            const int ncol, const int nlay, const int nbnd, const int ngpt,
            const int nflav, const int neta, const int npres, const int ntemp,
            const int nPlanckTemp,
            const Float* tlay,
            const Float* tlev,
            const Float* tsfc,
            const int sfc_lay,
            const Float* fmajor,
            const int* jeta,
            const Bool* tropo,
            const int* jtemp,
            const int* jpress,
            const int* gpoint_bands,
            const int* band_lims_gpt,
            const Float* pfracin,
            const Float temp_ref_min, const Float totplnk_delta,
            const Float* totplnk,
            const int* gpoint_flavor,
            Float* sfc_src,
            Float* lay_src,
            Float* lev_src_inc,
            Float* lev_src_dec,
            Float* sfc_src_jac)
    {
        Tuner_map& tunings = Tuner::get_map();

        const Float delta_Tsurf = Float(1.);

        const int block_gpt = 16;
        const int block_lay = 4;
        const int block_col = 2;

        const int grid_gpt = ngpt/block_gpt + (ngpt%block_gpt > 0);
        const int grid_lay = nlay/block_lay + (nlay%block_lay > 0);
        const int grid_col = ncol/block_col + (ncol%block_col > 0);

        dim3 grid_gpu(grid_gpt, grid_lay, grid_col);
        dim3 block_gpu(block_gpt, block_lay, block_col);
        
        if (tunings.count("Planck_source_kernel") == 0)
        {
            std::tie(grid_gpu, block_gpu) = tune_kernel(
                    "Planck_source_kernel",
                    dim3(ngpt, nlay, ncol),
                    {1, 2, 4},
                    {1, 2},
                    {1, 2, 4, 8, 16, 32, 48, 64, 96, 128, 256},
                    Planck_source_kernel,
                    ncol, nlay, nbnd, ngpt,
                    nflav, neta, npres, ntemp, nPlanckTemp,
                    tlay, tlev, tsfc, sfc_lay,
                    fmajor, jeta, tropo, jtemp,
                    jpress, gpoint_bands, band_lims_gpt,
                    pfracin, temp_ref_min, totplnk_delta,
                    totplnk, gpoint_flavor,
                    delta_Tsurf, sfc_src, lay_src,
                    lev_src_inc, lev_src_dec,
                    sfc_src_jac);
            
            tunings["Planck_source_kernel"].first = grid_gpu;
            tunings["Planck_source_kernel"].second = block_gpu;
        }
        else
        {
            grid_gpu = tunings["Planck_source_kernel"].first;
            block_gpu = tunings["Planck_source_kernel"].second;
        }

        Planck_source_kernel<<<grid_gpu, block_gpu>>>(
                ncol, nlay, nbnd, ngpt,
                nflav, neta, npres, ntemp, nPlanckTemp,
                tlay, tlev, tsfc, sfc_lay,
                fmajor, jeta, tropo, jtemp,
                jpress, gpoint_bands, band_lims_gpt,
                pfracin, temp_ref_min, totplnk_delta,
                totplnk, gpoint_flavor,
                delta_Tsurf,
                sfc_src, lay_src,
                lev_src_inc, lev_src_dec,
                sfc_src_jac);
    }
}
