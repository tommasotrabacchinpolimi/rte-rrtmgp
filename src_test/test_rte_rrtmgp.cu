#include "hip/hip_runtime.h"
/*
 * This file is a stand-alone executable developed for the
 * testing of the C++ interface to the RTE+RRTMGP radiation code.
 *
 * It is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * This software is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this software.  If not, see <http://www.gnu.org/licenses/>.
 */

#include <boost/algorithm/string.hpp>
#include <chrono>
#include <iomanip>
#include <hip/hip_runtime_api.h>


#include "Status.h"
#include "Netcdf_interface.h"
#include "Array.h"
#include "Radiation_solver.h"
#include "Gas_concs.h"
#include "Types.h"
#include "Mem_pool_gpu.h"


void read_and_set_vmr(
        const std::string& gas_name, const int n_col_x, const int n_col_y, const int n_lay,
        const Netcdf_handle& input_nc, Gas_concs& gas_concs)
{
    const std::string vmr_gas_name = "vmr_" + gas_name;

    if (input_nc.variable_exists(vmr_gas_name))
    {
        std::map<std::string, int> dims = input_nc.get_variable_dimensions(vmr_gas_name);
        const int n_dims = dims.size();

        if (n_dims == 0)
        {
            gas_concs.set_vmr(gas_name, input_nc.get_variable<Float>(vmr_gas_name));
        }
        else if (n_dims == 1)
        {
            if (dims.at("lay") == n_lay)
                gas_concs.set_vmr(gas_name,
                        Array<Float,1>(input_nc.get_variable<Float>(vmr_gas_name, {n_lay}), {n_lay}));
            else
                throw std::runtime_error("Illegal dimensions of gas \"" + gas_name + "\" in input");
        }
        else if (n_dims == 3)
        {
            if (dims.at("lay") == n_lay && dims.at("y") == n_col_y && dims.at("x") == n_col_x)
                gas_concs.set_vmr(gas_name,
                        Array<Float,2>(input_nc.get_variable<Float>(vmr_gas_name, {n_lay, n_col_y, n_col_x}), {n_col_x * n_col_y, n_lay}));
            else
                throw std::runtime_error("Illegal dimensions of gas \"" + gas_name + "\" in input");
        }
    }
    else
    {
        Status::print_warning("Gas \"" + gas_name + "\" not available in input file.");
    }
}


void configure_memory_pool(int nlays, int ncols, int nchunks, int ngpts, int nbnds)
{
    /* Heuristic way to set up memory pool queues */
    std::map<std::size_t, std::size_t> pool_queues = {
        {64, 20},
        {128, 20},
        {256, 10},
        {512, 10},
        {1024, 5},
        {2048, 5},
        {nchunks * ngpts * sizeof(Float), 16},
        {nchunks * nbnds * sizeof(Float), 16},
        {(nlays + 1) * ncols * sizeof(Float), 14},
        {(nlays + 1) * nchunks * sizeof(Float), 10},
        {(nlays + 1) * nchunks * nbnds * sizeof(Float), 4},
        {(nlays + 1) * nchunks * ngpts * sizeof(int)/2, 6},
        {(nlays + 1) * nchunks * ngpts * sizeof(Float), 18}
    };

    #ifdef GPU_MEM_POOL
    Memory_pool_gpu::init_instance(pool_queues);
    #endif
}

bool parse_command_line_options(
        std::map<std::string, std::pair<bool, std::string>>& command_line_options,
        int argc, char** argv)
{
    for (int i=1; i<argc; ++i)
    {
        std::string argument(argv[i]);
        boost::trim(argument);

        if (argument == "-h" || argument == "--help")
        {
            Status::print_message("Possible usage:");
            for (const auto& clo : command_line_options)
            {
                std::ostringstream ss;
                ss << std::left << std::setw(30) << ("--" + clo.first);
                ss << clo.second.second << std::endl;
                Status::print_message(ss);
            }
            return true;
        }

        // Check if option starts with --
        if (argument[0] != '-' || argument[1] != '-')
        {
            std::string error = argument + " is an illegal command line option.";
            throw std::runtime_error(error);
        }
        else
            argument.erase(0, 2);

        // Check if option has prefix no-
        bool enable = true;
        if (argument[0] == 'n' && argument[1] == 'o' && argument[2] == '-')
        {
            enable = false;
            argument.erase(0, 3);
        }

        if (command_line_options.find(argument) == command_line_options.end())
        {
            std::string error = argument + " is an illegal command line option.";
            throw std::runtime_error(error);
        }
        else
            command_line_options.at(argument).first = enable;
    }

    return false;
}


void print_command_line_options(
        const std::map<std::string, std::pair<bool, std::string>>& command_line_options)
{
    Status::print_message("Solver settings:");
    for (const auto& option : command_line_options)
    {
        std::ostringstream ss;
        ss << std::left << std::setw(20) << (option.first);
        ss << " = " << std::boolalpha << option.second.first << std::endl;
        Status::print_message(ss);
    }
}


void solve_radiation(int argc, char** argv)
{
    Status::print_message("###### Starting RTE+RRTMGP solver ######");

    ////// FLOW CONTROL SWITCHES //////
    // Parse the command line options.
    std::map<std::string, std::pair<bool, std::string>> command_line_options {
        {"shortwave"        , { true,  "Enable computation of shortwave radiation."}},
        {"longwave"         , { true,  "Enable computation of longwave radiation." }},
        {"fluxes"           , { true,  "Enable computation of fluxes."             }},
        {"cloud-optics"     , { false, "Enable cloud optics."                      }},
        {"output-optical"   , { false, "Enable output of optical properties."      }},
        {"output-bnd-fluxes", { false, "Enable output of band fluxes."             }},
        {"timings"          , { false, "Repeat computation 10x for run times."     }} };

    if (parse_command_line_options(command_line_options, argc, argv))
        return;

    const bool switch_shortwave         = command_line_options.at("shortwave"        ).first;
    const bool switch_longwave          = command_line_options.at("longwave"         ).first;
    const bool switch_fluxes            = command_line_options.at("fluxes"           ).first;
    const bool switch_cloud_optics      = command_line_options.at("cloud-optics"     ).first;
    const bool switch_output_optical    = command_line_options.at("output-optical"   ).first;
    const bool switch_output_bnd_fluxes = command_line_options.at("output-bnd-fluxes").first;
    const bool switch_timings           = command_line_options.at("timings"          ).first;

    // Print the options to the screen.
    print_command_line_options(command_line_options);


    ////// READ THE ATMOSPHERIC DATA //////
    Status::print_message("Reading atmospheric input data from NetCDF.");

    Netcdf_file input_nc("rte_rrtmgp_input.nc", Netcdf_mode::Read);

    const int n_col_x = input_nc.get_dimension_size("x");
    const int n_col_y = input_nc.get_dimension_size("y");
    const int n_lay = input_nc.get_dimension_size("lay");
    const int n_lev = input_nc.get_dimension_size("lev");
    const int n_col = n_col_x * n_col_y;
    // Read the atmospheric fields.
    Array<Float,2> p_lay(input_nc.get_variable<Float>("p_lay", {n_lay, n_col_y, n_col_x}), {n_col, n_lay});
    Array<Float,2> t_lay(input_nc.get_variable<Float>("t_lay", {n_lay, n_col_y, n_col_x}), {n_col, n_lay});
    Array<Float,2> p_lev(input_nc.get_variable<Float>("p_lev", {n_lev, n_col_y, n_col_x}), {n_col, n_lev});
    Array<Float,2> t_lev(input_nc.get_variable<Float>("t_lev", {n_lev, n_col_y, n_col_x}), {n_col, n_lev});

    // Fetch the col_dry in case present.
    Array<Float,2> col_dry;
    if (input_nc.variable_exists("col_dry"))
    {
        col_dry.set_dims({n_col, n_lay});
        col_dry = std::move(input_nc.get_variable<Float>("col_dry", {n_lay, n_col_y, n_col_x}));
    }

    // Create container for the gas concentrations and read gases.
    Gas_concs gas_concs;

    read_and_set_vmr("h2o", n_col_x, n_col_y, n_lay, input_nc, gas_concs);
    read_and_set_vmr("co2", n_col_x, n_col_y, n_lay, input_nc, gas_concs);
    read_and_set_vmr("o3" , n_col_x, n_col_y, n_lay, input_nc, gas_concs);
    read_and_set_vmr("n2o", n_col_x, n_col_y, n_lay, input_nc, gas_concs);
    read_and_set_vmr("co" , n_col_x, n_col_y, n_lay, input_nc, gas_concs);
    read_and_set_vmr("ch4", n_col_x, n_col_y, n_lay, input_nc, gas_concs);
    read_and_set_vmr("o2" , n_col_x, n_col_y, n_lay, input_nc, gas_concs);
    read_and_set_vmr("n2" , n_col_x, n_col_y, n_lay, input_nc, gas_concs);

    read_and_set_vmr("ccl4"   , n_col_x, n_col_y, n_lay, input_nc, gas_concs);
    read_and_set_vmr("cfc11"  , n_col_x, n_col_y, n_lay, input_nc, gas_concs);
    read_and_set_vmr("cfc12"  , n_col_x, n_col_y, n_lay, input_nc, gas_concs);
    read_and_set_vmr("cfc22"  , n_col_x, n_col_y, n_lay, input_nc, gas_concs);
    read_and_set_vmr("hfc143a", n_col_x, n_col_y, n_lay, input_nc, gas_concs);
    read_and_set_vmr("hfc125" , n_col_x, n_col_y, n_lay, input_nc, gas_concs);
    read_and_set_vmr("hfc23"  , n_col_x, n_col_y, n_lay, input_nc, gas_concs);
    read_and_set_vmr("hfc32"  , n_col_x, n_col_y, n_lay, input_nc, gas_concs);
    read_and_set_vmr("hfc134a", n_col_x, n_col_y, n_lay, input_nc, gas_concs);
    read_and_set_vmr("cf4"    , n_col_x, n_col_y, n_lay, input_nc, gas_concs);
    read_and_set_vmr("no2"    , n_col_x, n_col_y, n_lay, input_nc, gas_concs);

    Array<Float,2> lwp;
    Array<Float,2> iwp;
    Array<Float,2> rel;
    Array<Float,2> rei;

    if (switch_cloud_optics)
    {
        lwp.set_dims({n_col, n_lay});
        lwp = std::move(input_nc.get_variable<Float>("lwp", {n_lay, n_col_y, n_col_x}));

        iwp.set_dims({n_col, n_lay});
        iwp = std::move(input_nc.get_variable<Float>("iwp", {n_lay, n_col_y, n_col_x}));

        rel.set_dims({n_col, n_lay});
        rel = std::move(input_nc.get_variable<Float>("rel", {n_lay, n_col_y, n_col_x}));

        rei.set_dims({n_col, n_lay});
        rei = std::move(input_nc.get_variable<Float>("rei", {n_lay, n_col_y, n_col_x}));
    }


    ////// CREATE THE OUTPUT FILE //////
    // Create the general dimensions and arrays.
    Status::print_message("Preparing NetCDF output file.");

    Netcdf_file output_nc("rte_rrtmgp_output.nc", Netcdf_mode::Create);
    output_nc.add_dimension("x", n_col_x);
    output_nc.add_dimension("y", n_col_y);
    output_nc.add_dimension("lay", n_lay);
    output_nc.add_dimension("lev", n_lev);
    output_nc.add_dimension("pair", 2);

    auto nc_lay = output_nc.add_variable<Float>("p_lay", {"lay", "y", "x"});
    auto nc_lev = output_nc.add_variable<Float>("p_lev", {"lev", "y", "x"});

    nc_lay.insert(p_lay.v(), {0, 0, 0});
    nc_lev.insert(p_lev.v(), {0, 0, 0});

    int ngpts = 0;
    int nbnds = 0;
    if (switch_longwave)
    {
        Netcdf_file coef_nc_lw("coefficients_lw.nc", Netcdf_mode::Read);
        nbnds = std::max(coef_nc_lw.get_dimension_size("bnd"), nbnds);
        ngpts = std::max(coef_nc_lw.get_dimension_size("gpt"), ngpts);
    }
    if (switch_shortwave)
    {
        Netcdf_file coef_nc_sw("coefficients_sw.nc", Netcdf_mode::Read);
        nbnds = std::max(coef_nc_sw.get_dimension_size("bnd"), nbnds);
        ngpts = std::max(coef_nc_sw.get_dimension_size("gpt"), ngpts);
    }
    configure_memory_pool(n_lay, n_col, 512, ngpts, nbnds);


    ////// RUN THE LONGWAVE SOLVER //////
    if (switch_longwave)
    {
        // Initialize the solver.
        Status::print_message("Initializing the longwave solver.");

        Gas_concs_gpu gas_concs_gpu(gas_concs);
        
        Radiation_solver_longwave rad_lw(gas_concs_gpu, "coefficients_lw.nc", "cloud_coefficients_lw.nc");

        // Read the boundary conditions.
        const int n_bnd_lw = rad_lw.get_n_bnd_gpu();
        const int n_gpt_lw = rad_lw.get_n_gpt_gpu();

        Array<Float,2> emis_sfc(input_nc.get_variable<Float>("emis_sfc", {n_col_y, n_col_x, n_bnd_lw}), {n_bnd_lw, n_col});
        Array<Float,1> t_sfc(input_nc.get_variable<Float>("t_sfc", {n_col_y, n_col_x}), {n_col});

        // Create output arrays.
        Array_gpu<Float,3> lw_tau;
        Array_gpu<Float,3> lay_source;
        Array_gpu<Float,3> lev_source_inc;
        Array_gpu<Float,3> lev_source_dec;
        Array_gpu<Float,2> sfc_source;

        if (switch_output_optical)
        {
            lw_tau        .set_dims({n_col, n_lay, n_gpt_lw});
            lay_source    .set_dims({n_col, n_lay, n_gpt_lw});
            lev_source_inc.set_dims({n_col, n_lay, n_gpt_lw});
            lev_source_dec.set_dims({n_col, n_lay, n_gpt_lw});
            sfc_source    .set_dims({n_col, n_gpt_lw});
        }

        Array_gpu<Float,2> lw_flux_up;
        Array_gpu<Float,2> lw_flux_dn;
        Array_gpu<Float,2> lw_flux_net;

        if (switch_fluxes)
        {
            lw_flux_up .set_dims({n_col, n_lev});
            lw_flux_dn .set_dims({n_col, n_lev});
            lw_flux_net.set_dims({n_col, n_lev});
        }

        Array_gpu<Float,3> lw_bnd_flux_up;
        Array_gpu<Float,3> lw_bnd_flux_dn;
        Array_gpu<Float,3> lw_bnd_flux_net;

        if (switch_output_bnd_fluxes)
        {
            lw_bnd_flux_up .set_dims({n_col, n_lev, n_bnd_lw});
            lw_bnd_flux_dn .set_dims({n_col, n_lev, n_bnd_lw});
            lw_bnd_flux_net.set_dims({n_col, n_lev, n_bnd_lw});
        }


        // Solve the radiation.

        Status::print_message("Solving the longwave radiation.");

        auto run_solver = [&]()
        {
            Array_gpu<Float,2> p_lay_gpu(p_lay);
            Array_gpu<Float,2> p_lev_gpu(p_lev);
            Array_gpu<Float,2> t_lay_gpu(t_lay);
            Array_gpu<Float,2> t_lev_gpu(t_lev);
            Array_gpu<Float,2> col_dry_gpu(col_dry);
            Array_gpu<Float,1> t_sfc_gpu(t_sfc);
            Array_gpu<Float,2> emis_sfc_gpu(emis_sfc);
            Array_gpu<Float,2> lwp_gpu(lwp);
            Array_gpu<Float,2> iwp_gpu(iwp);
            Array_gpu<Float,2> rel_gpu(rel);
            Array_gpu<Float,2> rei_gpu(rei);

            hipDeviceSynchronize();
            hipEvent_t start;
            hipEvent_t stop;
            hipEventCreate(&start);
            hipEventCreate(&stop);

            hipEventRecord(start, 0);

            rad_lw.solve_gpu(
                    switch_fluxes,
                    switch_cloud_optics,
                    switch_output_optical,
                    switch_output_bnd_fluxes,
                    gas_concs_gpu,
                    p_lay_gpu, p_lev_gpu,
                    t_lay_gpu, t_lev_gpu,
                    col_dry_gpu,
                    t_sfc_gpu, emis_sfc_gpu,
                    lwp_gpu, iwp_gpu,
                    rel_gpu, rei_gpu,
                    lw_tau, lay_source, lev_source_inc, lev_source_dec, sfc_source,
                    lw_flux_up, lw_flux_dn, lw_flux_net,
                    lw_bnd_flux_up, lw_bnd_flux_dn, lw_bnd_flux_net);

            hipEventRecord(stop, 0);
            hipEventSynchronize(stop);
            float duration = 0.f;
            hipEventElapsedTime(&duration, start, stop);

            hipEventDestroy(start);
            hipEventDestroy(stop);

            Status::print_message("Duration longwave solver: " + std::to_string(duration) + " (ms)");
        };

        // Tuning step;
        run_solver();

        // Profiling step;
        hipProfilerStart();
        run_solver();
        hipProfilerStop();

        if (switch_timings)
        {
            constexpr int n_measures=10;
            for (int n=0; n<n_measures; ++n)
                run_solver();
        }

        //// Store the output.
        Status::print_message("Storing the longwave output.");
        Array<Float,3> lw_tau_cpu(lw_tau);
        Array<Float,3> lay_source_cpu(lay_source);
        Array<Float,2> sfc_source_cpu(sfc_source);
        Array<Float,3> lev_source_inc_cpu(lev_source_inc);
        Array<Float,3> lev_source_dec_cpu(lev_source_dec);
        Array<Float,2> lw_flux_up_cpu(lw_flux_up);
        Array<Float,2> lw_flux_dn_cpu(lw_flux_dn);
        Array<Float,2> lw_flux_net_cpu(lw_flux_net);
        Array<Float,3> lw_bnd_flux_up_cpu(lw_bnd_flux_up);
        Array<Float,3> lw_bnd_flux_dn_cpu(lw_bnd_flux_dn);
        Array<Float,3> lw_bnd_flux_net_cpu(lw_bnd_flux_net);

        output_nc.add_dimension("gpt_lw", n_gpt_lw);
        output_nc.add_dimension("band_lw", n_bnd_lw);

        auto nc_lw_band_lims_wvn = output_nc.add_variable<Float>("lw_band_lims_wvn", {"band_lw", "pair"});
        nc_lw_band_lims_wvn.insert(rad_lw.get_band_lims_wavenumber_gpu().v(), {0, 0});

        if (switch_output_optical)
        {
            auto nc_lw_band_lims_gpt = output_nc.add_variable<int>("lw_band_lims_gpt", {"band_lw", "pair"});
            nc_lw_band_lims_gpt.insert(rad_lw.get_band_lims_gpoint_gpu().v(), {0, 0});

            auto nc_lw_tau = output_nc.add_variable<Float>("lw_tau", {"gpt_lw", "lay", "y", "x"});
            nc_lw_tau.insert(lw_tau_cpu.v(), {0, 0, 0, 0});

            auto nc_lay_source     = output_nc.add_variable<Float>("lay_source"    , {"gpt_lw", "lay", "y", "x"});
            auto nc_lev_source_inc = output_nc.add_variable<Float>("lev_source_inc", {"gpt_lw", "lay", "y", "x"});
            auto nc_lev_source_dec = output_nc.add_variable<Float>("lev_source_dec", {"gpt_lw", "lay", "y", "x"});

            auto nc_sfc_source = output_nc.add_variable<Float>("sfc_source", {"gpt_lw", "y", "x"});

            nc_lay_source.insert    (lay_source_cpu.v()    , {0, 0, 0, 0});
            nc_lev_source_inc.insert(lev_source_inc_cpu.v(), {0, 0, 0, 0});
            nc_lev_source_dec.insert(lev_source_dec_cpu.v(), {0, 0, 0, 0});

            nc_sfc_source.insert(sfc_source_cpu.v(), {0, 0, 0});
        }

        if (switch_fluxes)
        {
            auto nc_lw_flux_up  = output_nc.add_variable<Float>("lw_flux_up" , {"lev", "y", "x"});
            auto nc_lw_flux_dn  = output_nc.add_variable<Float>("lw_flux_dn" , {"lev", "y", "x"});
            auto nc_lw_flux_net = output_nc.add_variable<Float>("lw_flux_net", {"lev", "y", "x"});

            nc_lw_flux_up .insert(lw_flux_up_cpu .v(), {0, 0, 0});
            nc_lw_flux_dn .insert(lw_flux_dn_cpu .v(), {0, 0, 0});
            nc_lw_flux_net.insert(lw_flux_net_cpu.v(), {0, 0, 0});

            if (switch_output_bnd_fluxes)
            {
                auto nc_lw_bnd_flux_up  = output_nc.add_variable<Float>("lw_bnd_flux_up" , {"band_lw", "lev", "y", "x"});
                auto nc_lw_bnd_flux_dn  = output_nc.add_variable<Float>("lw_bnd_flux_dn" , {"band_lw", "lev", "y", "x"});
                auto nc_lw_bnd_flux_net = output_nc.add_variable<Float>("lw_bnd_flux_net", {"band_lw", "lev", "y", "x"});

                nc_lw_bnd_flux_up .insert(lw_bnd_flux_up_cpu.v(), {0, 0, 0, 0});
                nc_lw_bnd_flux_dn .insert(lw_bnd_flux_dn_cpu.v(), {0, 0, 0, 0});
                nc_lw_bnd_flux_net.insert(lw_bnd_flux_net_cpu.v(), {0, 0, 0, 0});
            }
        }
    }


    ////// RUN THE SHORTWAVE SOLVER //////
    if (switch_shortwave)
    {
        // Initialize the solver.
        Status::print_message("Initializing the shortwave solver.");


        Gas_concs_gpu gas_concs_gpu(gas_concs);
        Radiation_solver_shortwave rad_sw(gas_concs_gpu, "coefficients_sw.nc", "cloud_coefficients_sw.nc");

        // Read the boundary conditions.
        const int n_bnd_sw = rad_sw.get_n_bnd_gpu();
        const int n_gpt_sw = rad_sw.get_n_gpt_gpu();

        Array<Float,1> mu0(input_nc.get_variable<Float>("mu0", {n_col_y, n_col_x}), {n_col});
        Array<Float,2> sfc_alb_dir(input_nc.get_variable<Float>("sfc_alb_dir", {n_col_y, n_col_x, n_bnd_sw}), {n_bnd_sw, n_col});
        Array<Float,2> sfc_alb_dif(input_nc.get_variable<Float>("sfc_alb_dif", {n_col_y, n_col_x, n_bnd_sw}), {n_bnd_sw, n_col});

        Array<Float,1> tsi_scaling({n_col});
        if (input_nc.variable_exists("tsi"))
        {
            Array<Float,1> tsi(input_nc.get_variable<Float>("tsi", {n_col_y, n_col_x}), {n_col});
            const Float tsi_ref = rad_sw.get_tsi_gpu();
            for (int icol=1; icol<=n_col; ++icol)
                tsi_scaling({icol}) = tsi({icol}) / tsi_ref;
        }
        else
        {
            for (int icol=1; icol<=n_col; ++icol)
                tsi_scaling({icol}) = Float(1.);
        }

        // Create output arrays.
        Array_gpu<Float,3> sw_tau;
        Array_gpu<Float,3> ssa;
        Array_gpu<Float,3> g;
        Array_gpu<Float,2> toa_source;

        if (switch_output_optical)
        {
            sw_tau    .set_dims({n_col, n_lay, n_gpt_sw});
            ssa       .set_dims({n_col, n_lay, n_gpt_sw});
            g         .set_dims({n_col, n_lay, n_gpt_sw});
            toa_source.set_dims({n_col, n_gpt_sw});
        }

        Array_gpu<Float,2> sw_flux_up;
        Array_gpu<Float,2> sw_flux_dn;
        Array_gpu<Float,2> sw_flux_dn_dir;
        Array_gpu<Float,2> sw_flux_net;

        if (switch_fluxes)
        {
            sw_flux_up    .set_dims({n_col, n_lev});
            sw_flux_dn    .set_dims({n_col, n_lev});
            sw_flux_dn_dir.set_dims({n_col, n_lev});
            sw_flux_net   .set_dims({n_col, n_lev});
        }

        Array_gpu<Float,3> sw_bnd_flux_up;
        Array_gpu<Float,3> sw_bnd_flux_dn;
        Array_gpu<Float,3> sw_bnd_flux_dn_dir;
        Array_gpu<Float,3> sw_bnd_flux_net;

        if (switch_output_bnd_fluxes)
        {
            sw_bnd_flux_up    .set_dims({n_col, n_lev, n_bnd_sw});
            sw_bnd_flux_dn    .set_dims({n_col, n_lev, n_bnd_sw});
            sw_bnd_flux_dn_dir.set_dims({n_col, n_lev, n_bnd_sw});
            sw_bnd_flux_net   .set_dims({n_col, n_lev, n_bnd_sw});
        }


        // Solve the radiation.
        Status::print_message("Solving the shortwave radiation.");

        auto run_solver = [&]()
        {
            Array_gpu<Float,2> p_lay_gpu(p_lay);
            Array_gpu<Float,2> p_lev_gpu(p_lev);
            Array_gpu<Float,2> t_lay_gpu(t_lay);
            Array_gpu<Float,2> t_lev_gpu(t_lev);
            Array_gpu<Float,2> col_dry_gpu(col_dry);
            Array_gpu<Float,2> sfc_alb_dir_gpu(sfc_alb_dir);
            Array_gpu<Float,2> sfc_alb_dif_gpu(sfc_alb_dif);
            Array_gpu<Float,1> tsi_scaling_gpu(tsi_scaling);
            Array_gpu<Float,1> mu0_gpu(mu0);
            Array_gpu<Float,2> lwp_gpu(lwp);
            Array_gpu<Float,2> iwp_gpu(iwp);
            Array_gpu<Float,2> rel_gpu(rel);
            Array_gpu<Float,2> rei_gpu(rei);

            hipDeviceSynchronize();
            hipEvent_t start;
            hipEvent_t stop;
            hipEventCreate(&start);
            hipEventCreate(&stop);

            hipEventRecord(start, 0);

            rad_sw.solve_gpu(
                    switch_fluxes,
                    switch_cloud_optics,
                    switch_output_optical,
                    switch_output_bnd_fluxes,
                    gas_concs_gpu,
                    p_lay_gpu, p_lev_gpu,
                    t_lay_gpu, t_lev_gpu,
                    col_dry_gpu,
                    sfc_alb_dir_gpu, sfc_alb_dif_gpu,
                    tsi_scaling_gpu, mu0_gpu,
                    lwp_gpu, iwp_gpu,
                    rel_gpu, rei_gpu,
                    sw_tau, ssa, g,
                    toa_source,
                    sw_flux_up, sw_flux_dn,
                    sw_flux_dn_dir, sw_flux_net,
                    sw_bnd_flux_up, sw_bnd_flux_dn,
                    sw_bnd_flux_dn_dir, sw_bnd_flux_net);

            hipEventRecord(stop, 0);
            hipEventSynchronize(stop);
            float duration = 0.f;
            hipEventElapsedTime(&duration, start, stop);

            hipEventDestroy(start);
            hipEventDestroy(stop);

            Status::print_message("Duration shortwave solver: " + std::to_string(duration) + " (ms)");
        };

        // Tuning step;
        run_solver();

        // Profiling step;
        hipProfilerStart();
        run_solver();
        hipProfilerStop();

        if (switch_timings)
        {   
            constexpr int n_measures=10;
            for (int n=0; n<n_measures; ++n)
                run_solver();
        }

        // Store the output.
        Status::print_message("Storing the shortwave output.");
        Array<Float,3> sw_tau_cpu(sw_tau);
        Array<Float,3> ssa_cpu(ssa);
        Array<Float,3> g_cpu(g);
        Array<Float,2> toa_source_cpu(toa_source);
        Array<Float,2> sw_flux_up_cpu(sw_flux_up);
        Array<Float,2> sw_flux_dn_cpu(sw_flux_dn);
        Array<Float,2> sw_flux_dn_dir_cpu(sw_flux_dn_dir);
        Array<Float,2> sw_flux_net_cpu(sw_flux_net);
        Array<Float,3> sw_bnd_flux_up_cpu(sw_bnd_flux_up);
        Array<Float,3> sw_bnd_flux_dn_cpu(sw_bnd_flux_dn);
        Array<Float,3> sw_bnd_flux_dn_dir_cpu(sw_bnd_flux_dn_dir);
        Array<Float,3> sw_bnd_flux_net_cpu(sw_bnd_flux_net);

        output_nc.add_dimension("gpt_sw", n_gpt_sw);
        output_nc.add_dimension("band_sw", n_bnd_sw);

        auto nc_sw_band_lims_wvn = output_nc.add_variable<Float>("sw_band_lims_wvn", {"band_sw", "pair"});
        nc_sw_band_lims_wvn.insert(rad_sw.get_band_lims_wavenumber_gpu().v(), {0, 0});

        if (switch_output_optical)
        {
            auto nc_sw_band_lims_gpt = output_nc.add_variable<int>("sw_band_lims_gpt", {"band_sw", "pair"});
            nc_sw_band_lims_gpt.insert(rad_sw.get_band_lims_gpoint_gpu().v(), {0, 0});

            auto nc_sw_tau = output_nc.add_variable<Float>("sw_tau", {"gpt_sw", "lay", "y", "x"});
            auto nc_ssa    = output_nc.add_variable<Float>("ssa"   , {"gpt_sw", "lay", "y", "x"});
            auto nc_g      = output_nc.add_variable<Float>("g"     , {"gpt_sw", "lay", "y", "x"});

            nc_sw_tau.insert(sw_tau_cpu.v(), {0, 0, 0, 0});
            nc_ssa   .insert(ssa_cpu   .v(), {0, 0, 0, 0});
            nc_g     .insert(g_cpu     .v(), {0, 0, 0, 0});

            auto nc_toa_source = output_nc.add_variable<Float>("toa_source", {"gpt_sw", "y", "x"});
            nc_toa_source.insert(toa_source_cpu.v(), {0, 0, 0});
        }

        if (switch_fluxes)
        {
            auto nc_sw_flux_up     = output_nc.add_variable<Float>("sw_flux_up"    , {"lev", "y", "x"});
            auto nc_sw_flux_dn     = output_nc.add_variable<Float>("sw_flux_dn"    , {"lev", "y", "x"});
            auto nc_sw_flux_dn_dir = output_nc.add_variable<Float>("sw_flux_dn_dir", {"lev", "y", "x"});
            auto nc_sw_flux_net    = output_nc.add_variable<Float>("sw_flux_net"   , {"lev", "y", "x"});

            nc_sw_flux_up    .insert(sw_flux_up_cpu    .v(), {0, 0, 0});
            nc_sw_flux_dn    .insert(sw_flux_dn_cpu    .v(), {0, 0, 0});
            nc_sw_flux_dn_dir.insert(sw_flux_dn_dir_cpu.v(), {0, 0, 0});
            nc_sw_flux_net   .insert(sw_flux_net_cpu   .v(), {0, 0, 0});

            if (switch_output_bnd_fluxes)
            {
                auto nc_sw_bnd_flux_up     = output_nc.add_variable<Float>("sw_bnd_flux_up"    , {"band_sw", "lev", "y", "x"});
                auto nc_sw_bnd_flux_dn     = output_nc.add_variable<Float>("sw_bnd_flux_dn"    , {"band_sw", "lev", "y", "x"});
                auto nc_sw_bnd_flux_dn_dir = output_nc.add_variable<Float>("sw_bnd_flux_dn_dir", {"band_sw", "lev", "y", "x"});
                auto nc_sw_bnd_flux_net    = output_nc.add_variable<Float>("sw_bnd_flux_net"   , {"band_sw", "lev", "y", "x"});

                nc_sw_bnd_flux_up    .insert(sw_bnd_flux_up_cpu    .v(), {0, 0, 0, 0});
                nc_sw_bnd_flux_dn    .insert(sw_bnd_flux_dn_cpu    .v(), {0, 0, 0, 0});
                nc_sw_bnd_flux_dn_dir.insert(sw_bnd_flux_dn_dir_cpu.v(), {0, 0, 0, 0});
                nc_sw_bnd_flux_net   .insert(sw_bnd_flux_net_cpu   .v(), {0, 0, 0, 0});
            }
        }
    }

    Status::print_message("###### Finished RTE+RRTMGP cuda solver ######");
}


int main(int argc, char** argv)
{
    try
    {
        solve_radiation(argc, argv);
    }

    // Catch any exceptions and return 1.
    catch (const std::exception& e)
    {
        std::string error = "EXCEPTION: " + std::string(e.what());
        Status::print_message(error);
        return 1;
    }
    catch (...)
    {
        Status::print_message("UNHANDLED EXCEPTION!");
        return 1;
    }

    // Return 0 in case of normal exit.
    return 0;
}
